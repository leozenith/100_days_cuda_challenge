#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void clear_data(float* data, int N){
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N){
        data[idx]=(float)0;
    }
}

__global__ 
void coalescedKernel(float* input, float* output, int N){
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N){
        output[idx] = input[idx] * 2.0f;
    }
}

__global__ 
void nonCoalescedKernel(float* input, float* output, int stride, int N){
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N){
        const int i = (idx * stride) % N;
        output[i] = input[i] * 2.0f;
    }
}

int main() {
    int N = 1 << 40; // 1M elements
    size_t size = N * sizeof(float);
    hipError_t err;

    // Allocate host memory
    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    // Initialize host array
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    // Allocate device memory
    float *d_input, *d_output, *d_dummy;
    err = hipMalloc((void**)&d_input, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
    err = hipMalloc((void**)&d_output, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
    err = hipMalloc((void**)&d_dummy, size); // For cache flushing
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }

    // Configure kernel execution
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    int stride = 2;

    // Benchmark setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float millisecondsCoalesced = 0, millisecondsNonCoalesced = 0;

    //warmup
    for(int i=0; i < 10; i++)coalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    clear_data<<<blocksPerGrid, threadsPerBlock>>>(d_dummy, N);
    hipDeviceSynchronize();

    // Benchmark coalesced kernel
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice); // Fresh input
    hipMemset(d_output, 0, size); // Fresh output
    hipEventRecord(start);
    coalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecondsCoalesced, start, stop);

    // Flush cache with dummy kernel
    clear_data<<<blocksPerGrid, threadsPerBlock>>>(d_dummy, N);
    hipDeviceSynchronize();

    // Benchmark non-coalesced kernel
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice); // Reset input
    hipMemset(d_output, 0, size); // Reset output
    hipEventRecord(start);
    nonCoalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N, stride);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecondsNonCoalesced, start, stop);

    // Print benchmark results
    printf("Coalesced Kernel Time:    %f ms\n", millisecondsCoalesced);
    printf("Non-Coalesced Kernel Time (stride=%d): %f ms\n", stride, millisecondsNonCoalesced);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_dummy);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
